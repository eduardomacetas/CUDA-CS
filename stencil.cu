
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

#define N 100000

#define BLOCK_SIZE 32

#define RADIUS 3


void randomsInt(int *f)
{
    for(int i=0;i<N;++i)
        f[i]=rand() % 10 + 1;
}


__global__ void stencil_1d(int *in, int *out) {
     __shared__ int temp[BLOCK_SIZE + 2 * RADIUS];
     int gindex = threadIdx.x + blockIdx.x * blockDim.x;
     int lindex = threadIdx.x + RADIUS;
     // Read input elements into shared memory
     temp[lindex] = in[gindex];
     if (threadIdx.x < RADIUS) {
     	temp[lindex - RADIUS] = in[gindex - RADIUS];
     	temp[lindex + BLOCK_SIZE] = in[gindex + BLOCK_SIZE];
     }

    // Synchronize (ensure all the data is available)
    __syncthreads();

    // Apply the stencil
    int result = 0;
    for (int offset = -RADIUS ; offset <= RADIUS ; offset++)
     result += temp[lindex + offset];
    // Store the result
    out[gindex] = result;
}

void serial (int *in, int *out){
	
	for(int i=RADIUS;i<N-RADIUS;++i){
	     int suma=0;
	     for(int j=i-RADIUS;j<=i+RADIUS;++j)
		 suma+=in[j];
	    out[i]=suma;
	}

}


void print(int * a){
	for (int i=0;i<N;++i){
	    cout<<a[i]<<"\t";
        }
	cout<<endl;
}


int main()
{
	srand (time(NULL));
	int *a, *c, *outserial;

	int *d_a, *d_c;

	int size = N * sizeof(int);
	
	hipMalloc((void **)&d_a,size);

	hipMalloc((void **)&d_c,size);


	a = (int *)malloc(size); randomsInt(a);
	c = (int *)malloc(size);
	outserial=(int *)malloc(size);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);


	hipEvent_t start,stop;

	hipEventCreate(&start);
	hipEventCreate(&stop);
	
	hipEventRecord(start);
        stencil_1d<<<(N + BLOCK_SIZE-1) / BLOCK_SIZE,BLOCK_SIZE>>>(d_a,d_c);

	hipEventRecord(stop);

	float timeCUDA;
	hipEventElapsedTime(&timeCUDA, start,stop);
	cout<<timeCUDA<<endl;


	hipMemcpy(c,d_c, size, hipMemcpyDeviceToHost);

	
        hipFree(d_a);hipFree(d_c);


	clock_t tSerial=clock();

	
	serial(a,outserial);

	tSerial=clock()-tSerial;
	cout<<"serial: "<< (double) tSerial/CLOCKS_PER_SEC <<endl;


        /*cout<<"----A----"<<endl;
	print (a);
	cout<<"----c----"<<endl;
	print(c);


	cout<<"----SERIAL----"<<endl;
	print(outserial);
*/
	free(a); free(c);


	return 0;
}
