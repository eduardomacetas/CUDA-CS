
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

#define row 4
#define column 4

#define threadsPB 16


void randomsInt(double **& matrix)
{
    for(int i=0;i<row;++i){
	for(int j=0;j<column;++j)
            matrix[i][j]=rand() % 10 + 1;
    }
}

void createMatrixHostCUDA(double**& host, double**& device, double **& aux, int size, int r, int c ){
    host = (double **)malloc(r*sizeof(double*));
    host[0]=(double *)malloc(size);


    aux =(double **)malloc(r*sizeof(double*));
    hipMalloc((void **)&aux[0],size);
 
    hipMalloc((void **)&device,r*sizeof(double*));
//    cudaMalloc((void **)&(device[0]),size);

    for (int i=1; i<r;++i){
	host[i]=host[i-1]+c;
	aux[i]=aux[i-1]+c;

    }

    hipMemcpy(device, aux, r*sizeof(double*), hipMemcpyHostToDevice);
    

}


__global__ void MatAdd(double ** A, double ** B,double ** C) {
     
     //printf("asdfas ");
     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
	C[i][j]= A[i][j] + B[i][j];

//       printf("i: %i\t j: %i\n" ,i,j );
       printf("1: %f \n " ,C[i][j] );
//       printf("2: %f \n" ,A[i][j] );
     
	
     }
     
}


void print(double ** a){
	for(int i=0;i<row;++i){
	    for(int j=0;j<column;++j)
            cout<<a[i][j]<<'\t';
	cout<<endl;
    }
	cout<<endl;
}


int main()
{
	srand (time(NULL));
	double **a, **b, **c;

	double **d_a, **d_b, **d_c;


	double **a_aux, **b_aux, **c_aux;
	
	int size = row* column * sizeof(double*);

	
	createMatrixHostCUDA(a,d_a,a_aux,size,row,column);
	createMatrixHostCUDA(b,d_b,b_aux,size,row,column);
	createMatrixHostCUDA(c,d_c,c_aux,size,row,column);

	randomsInt(a);randomsInt(b);
	

	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);
	hipMemcpy(b_aux[0], b[0], size, hipMemcpyHostToDevice);
	
	dim3 threadPerBlock(threadsPB, threadsPB);
	dim3 blockPerGrid((row+threadPerBlock.x-1)/threadPerBlock.x,(column+threadPerBlock.y-1)/threadPerBlock.y);
        
        MatAdd<<<blockPerGrid,threadPerBlock>>>(d_a,d_b,d_c);

	
	hipMemcpy(c[0],c_aux[0], size, hipMemcpyDeviceToHost);
	

	hipFree(d_a);hipFree(d_b);hipFree(d_c);
	hipFree(a_aux[0]);hipFree(b_aux[0]);hipFree(c_aux[0]);


        cout<<"----A----"<<endl;
	print (a);

        cout<<"----B----"<<endl;
	print (b);

	cout<<"----c----"<<endl;
	print(c);


	free(a); free(b); free(c);


	return 0;
}


