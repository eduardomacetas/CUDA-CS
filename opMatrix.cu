
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */
#include <math.h>

using namespace std;

#define row 4
#define column 4

#define threadsPB 16


void randomsInt(double **& matrix)
{
    for(int i=0;i<row;++i){
	for(int j=0;j<column;++j)
            matrix[i][j]=rand() % 10 + 1;
    }
}

void createMatrixHostCUDA(double**& host, double**& device, double **& aux, int size, int r, int c ){
    host = (double **)malloc(r*sizeof(double*));
    host[0]=(double *)malloc(size);


    aux =(double **)malloc(r*sizeof(double*));
    hipMalloc((void **)&aux[0],size);
 
    hipMalloc((void **)&device,r*sizeof(double*));
//    cudaMalloc((void **)&(device[0]),size);

    for (int i=1; i<r;++i){
	host[i]=host[i-1]+c;
	aux[i]=aux[i-1]+c;

    }

    hipMemcpy(device, aux, r*sizeof(double*), hipMemcpyHostToDevice);
    

}


__global__ void sigmoid(double ** A,double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
	C[i][j]= 1.0/(1+exp(-A[i][j]));

//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     
	
     }
     
}

__global__ void sigmoidGradient(double ** A, double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
	C[i][j]= (1.0/(1+exp(-A[i][j])))*(1-(1.0/(1+exp(-A[i][j]))));
	
     }
     
}

__global__ void scalarMult(double ** A, double s,double ** C) {
     
     //printf("asdfas ");

     int i= threadIdx.x  + blockIdx.x * blockDim.x;
     int j = threadIdx.y + blockIdx.y * blockDim.y;

     if (i <row && j <column){
	C[i][j]= A[i][j]* s;

//       printf("i: %i\t j: %i\n" ,i,j );
//       printf("2: %f \n" ,A[i][j] );
     
	
     }
     
}


void print(double ** a){
	for(int i=0;i<row;++i){
	    for(int j=0;j<column;++j)
            cout<<a[i][j]<<'\t';
	cout<<endl;
    }
	cout<<endl;
}


int main()
{
	srand (time(NULL));
	double **a, **c;

	double **d_a, **d_c;


	double **a_aux, **c_aux;
	
	int size = row* column * sizeof(double*);

	
	createMatrixHostCUDA(a,d_a,a_aux,size,row,column);

	createMatrixHostCUDA(c,d_c,c_aux,size,row,column);

	randomsInt(a);
	

	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);

	
	dim3 threadPerBlock(threadsPB, threadsPB);
	dim3 blockPerGrid((row+threadPerBlock.x-1)/threadPerBlock.x,(column+threadPerBlock.y-1)/threadPerBlock.y);
        
        scalarMult<<<blockPerGrid,threadPerBlock>>>(d_a,2,d_c);

	
	hipMemcpy(c[0],c_aux[0], size, hipMemcpyDeviceToHost);
	

	hipFree(d_a);hipFree(d_c);
	hipFree(a_aux[0]);hipFree(c_aux[0]);


        cout<<"----A----"<<endl;
	print (a);

        
	cout<<"----c----"<<endl;
	print(c);


	free(a); free(c);


	return 0;
}


