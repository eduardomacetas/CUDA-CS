
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

#define row 5
#define column 5

#define threadsPB 4


void randomsInt(double **& matrix)
{
    for(int i=0;i<row;++i){
	for(int j=0;j<column;++j)
            matrix[i][j]=rand() % 10 + 1;
    }
}

void createMatrixHostCUDA(double**& host, double**& device, double **& aux, int size, int r, int c ){
    host = (double **)malloc(r*sizeof(double*));
    host[0]=(double *)malloc(size);


    aux =(double **)malloc(r*sizeof(double*));
    hipMalloc((void **)&aux[0],size);
 
    hipMalloc((void **)&device,r*sizeof(double*));
//    cudaMalloc((void **)&(device[0]),size);

    for (int i=1; i<r;++i){
	host[i]=host[i-1]+c;
	aux[i]=aux[i-1]+c;

    }

    hipMemcpy(device, aux, r*sizeof(double*), hipMemcpyHostToDevice);
    

}


__global__ void MatAdd(double ** A, double ** B,double ** C) {
     

     int gIndexX= threadIdx.x  + blockIdx.x * blockDim.x;
     int gIndexY = threadIdx.y + blockIdx.y * blockDim.y;
     
     

     int lIndexX= threadIdx.x;
     int lIndexY = threadIdx.y;


     __shared__ int tempA[threadsPB][threadsPB], tempB[threadsPB][threadsPB];
   

     // Read input elements into shared memory

     if (gIndexX < row && gIndexY < column) {

	    tempA[lIndexX][lIndexY] = A[gIndexX][gIndexY];
	    tempB[lIndexX][lIndexY] = B[gIndexX][gIndexY];

	
     }

    // Synchronize (ensure all the data is available)
    __syncthreads();


    // Apply the stencil
  

    if (gIndexX <row && gIndexY <column){
	double count=0;
	for (int i=0;i<threadsPB;++i){
	    count+= tempA[lIndexX][i] * tempB[i][lIndexY];
	}

	C[lIndexX][lIndexY]= count;
	
    }



    /*if (gIndexX <row && gIndexY <column){
	double count=0;
	for (int i=0;i<row;++i){
	    count+= A[gIndexX][i] * B[i][gIndexY];
	}

	C[gIndexX][gIndexY]= count;
	
    }*/

}


void print(double ** a){
	for(int i=0;i<row;++i){
	    for(int j=0;j<column;++j)
            cout<<a[i][j]<<'\t';
	cout<<endl;
    }
	cout<<endl;
}


int main()
{
	srand (time(NULL));
	double **a, **b, **c;

	double **d_a, **d_b, **d_c;


	double **a_aux, **b_aux, **c_aux;
	

	
	int size = row* column * sizeof(double*);

	
	createMatrixHostCUDA(a,d_a,a_aux,size,row,column);
	createMatrixHostCUDA(b,d_b,b_aux,size,row,column);
	createMatrixHostCUDA(c,d_c,c_aux,size,row,column);


	randomsInt(a);randomsInt(b);
	

	hipMemcpy(a_aux[0], a[0], size, hipMemcpyHostToDevice);
	hipMemcpy(b_aux[0], b[0], size, hipMemcpyHostToDevice);
	
	dim3 threadPerBlock(threadsPB, threadsPB);
	dim3 blockPerGrid((row+threadPerBlock.x-1)/threadPerBlock.x,(column+threadPerBlock.y-1)/threadPerBlock.y);
        
        MatAdd<<<blockPerGrid,threadPerBlock>>>(d_a,d_b,d_c);

	
	hipMemcpy(c[0],c_aux[0], size, hipMemcpyDeviceToHost);
	

	hipFree(d_a);hipFree(d_b);hipFree(d_c);
	hipFree(a_aux[0]);hipFree(b_aux[0]);hipFree(c_aux[0]);


        cout<<"----A----"<<endl;
	print (a);

        cout<<"----B----"<<endl;
	print (b);

	cout<<"----c----"<<endl;
	print(c);


	free(a); free(b); free(c);


	return 0;
}


