
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

#define N 100

#define M 10

void randomsInt(int *f)
{
    for(int i=0;i<N;++i)
        f[i]=rand() % 100 + 1;
}

__global__ void add(int *a, int *b, int *c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N)
        c[index] = a[index] + b[index];
}

int main()
{
	srand (time(NULL));
	int *a, *b, *c;
	int *d_a, *d_b, *d_c;
	int size = N * sizeof(int);


	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);

	
	a = (int *)malloc(size); randomsInt(a);
	b = (int *)malloc(size); randomsInt(b);
	c = (int *)malloc(size); 


	hipMemcpy(d_a,a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        add<<<(N + M-1) / M,M>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c, size, hipMemcpyDeviceToHost);

	
        hipFree(d_a);hipFree(d_b);hipFree(d_c);

        for (int i=0;i<N;++i){
            cout<<a[i]<<"\t";
            cout<<b[i]<<"\t";
            cout<<c[i]<<"\n";
        }


	free(a); free(b); free(c);


	return 0;
}
