
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>     /* srand, rand */
#include <time.h>       /* time */

using namespace std;

#define filas 10

#define columnas 10

#define N filas*columnas

#define M filas*columnas


void randomsInt(int *f)
{
    for(int i=0;i<filas;++i)
	for(int j=0;j<columnas;++j)
            f[(i*filas)+j]=rand() % 10 + 1;
}


__global__ void add(int* a, int* b, int* c)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < N){
	int count=0;
	for (int i=0;i<filas;++i){
        	count += a[(((int)(index/filas))*filas)+i] * b[(i*filas)+(index%filas)];
	}
	c[index]=count;
   }
}

void printMatrix(int * a){

	for (int i=0;i<filas;++i){
	    for (int j=0;j<columnas;++j)
  	          cout<<a[(i*filas)+j]<<"\t";
//            cout<<b[i]<<"\t";
            //cout<<c[i]<<"\n";
	    cout<<endl;
        }
}



int main()
{
	cout<<(int)(5/filas)<<endl;
	srand (time(NULL));
	int *a, *b, *c;

	int *d_a, *d_b, *d_c;

	int size = filas*columnas * sizeof(int);
	
	hipMalloc((void **)&d_a,size);
	hipMalloc((void **)&d_b,size);
	hipMalloc((void **)&d_c,size);


	a = (int *)malloc(size); randomsInt(a);
	b = (int *)malloc(size); randomsInt(b);
	c = (int *)malloc(size);


	hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

        add<<<(N + M-1) / M,M>>>(d_a,d_b,d_c);

	hipMemcpy(c,d_c, size, hipMemcpyDeviceToHost);

	
        hipFree(d_a);hipFree(d_b);hipFree(d_c);

        cout<<"----A----"<<endl;
	printMatrix (a);
cout<<"----B----"<<endl;

	printMatrix (b);
cout<<"----c----"<<endl;
	printMatrix (c);

	//free(a); free(b); free(c);


	return 0;
}
